// vector_add.cu

#include <hip/hip_runtime.h>
#include <stdexcept>

__global__
void add_kernel(const float* a, const float* b, float* result, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        result[i] = a[i] + b[i];
    }
}

void vector_add_cuda(const float* a, const float* b, float* result, int n) {
    float *d_a, *d_b, *d_result;
    size_t size = n * sizeof(float);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_result, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    add_kernel<<<blocks, threads>>>(d_a, d_b, d_result, n);

    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
}
